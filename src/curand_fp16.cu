#include "hip/hip_runtime.h"
#include <curand_fp16/curand_fp16.hpp>
#include <stdexcept>

namespace {
constexpr unsigned block_size = 512;
template <class T>
__global__ void status_init_kernel(
		T* const status_ptr,
		const std::uint64_t seed
		) {
	const auto tid = blockDim.x * blockIdx.x + threadIdx.x;
	hiprand_init(seed, tid, 0, status_ptr + tid);
}

template <class T>
struct size_of{int value = 0;};
template <> struct size_of<ushort1> {static const int value = 2;};
template <> struct size_of<uint1  > {static const int value = 4;};
template <> struct size_of<ulong2 > {static const int value = 16;};
template <> struct size_of<half   > {static const int value = 2;};
template <> struct size_of<half2  > {static const int value = 4;};

template <class RNG_T>
__global__ void generate_kernel(
		half* const array_ptr,
		RNG_T* const status_ptr,
		const std::size_t size
		) {
	const auto batch_size = size_of<ulong2>::value / size_of<half>::value;
	const auto tid = blockDim.x * blockIdx.x + threadIdx.x;
	auto curand_gen = *(status_ptr + tid);
	for (unsigned i = tid * batch_size; i < size; i += batch_size * gridDim.x * blockDim.x) {
		const auto res = size - i;
		if (res < batch_size) {
			// TODO
		} else {
			// block gen
			union {
				half   h1[size_of<ulong2>::value / size_of<half >::value];
				half2  h2[size_of<ulong2>::value / size_of<half2>::value];
				ulong2 ul2;
			} batch_block;

			for (unsigned j = 0; j < size_of<ulong2>::value / size_of<uint1>::value; j++) {
				union {
					ushort1 us[size_of<uint1>::value / size_of<ushort1>::value];
					uint1 ui1;
				} rand_batch_block;
				rand_batch_block.ui1.x = hiprand(&curand_gen);
				for (unsigned k = 0; k < size_of<uint1>::value / size_of<half>::value; k++) {
					const auto us = rand_batch_block.us[k];
					const auto v  = __float2half(static_cast<float>(us.x) / static_cast<float>(0xffff));

					batch_block.h1[k + j * size_of<uint1>::value / size_of<half>::value] = v;
				}
			}
			*reinterpret_cast<ulong2*>(array_ptr + i) = batch_block.ul2;
		}
	}
	*(status_ptr + tid) = curand_gen;
}
} // noname namespace

void mtk::curand_fp16::create(generator_t &gen, const hiprandRngType_t rng_type) {
	// set cuda stream
	gen.hip_stream = 0;
	// get num sm
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	gen.num_sm = prop.multiProcessorCount;

	// calculate grid_size
	gen.num_threads = gen.num_sm * 16 * block_size;

	// set algo
	gen.rng_type = rng_type;

	// set generator
	unsigned state_struct_size = 0;
	switch (rng_type) {
#define CASE_RNG_TYPE(rng) case rng: state_struct_size = sizeof(typename mtk::curand_fp16::curand_status_t<rng>::type);break
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A        );
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW          );
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10   );
		default:
			throw std::runtime_error("Unknown pseudo rand algorithm");
#undef CASE_RNG_TYPE
	}
	const auto stat = hipMalloc(&gen.status_ptr, state_struct_size * gen.num_threads);
	if (stat != hipSuccess) {
		throw std::runtime_error("[curand_fp16 error] : " + std::string(hipGetErrorString(stat)) + " @" + __func__);
	}
}

void mtk::curand_fp16::set_seed(generator_t &gen, const std::uint64_t seed) {
	switch (gen.rng_type) {
#define CASE_RNG_TYPE(rng) case rng: status_init_kernel<typename mtk::curand_fp16::curand_status_t<rng>::type>\
		<<<gen.num_threads / block_size, block_size, 0, gen.hip_stream>>>\
		(reinterpret_cast<typename mtk::curand_fp16::curand_status_t<rng>::type*>(gen.status_ptr), seed);break
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A        );
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW          );
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10   );
		default:
			throw std::runtime_error("Unknown pseudo rand algorithm");
#undef CASE_RNG_TYPE
	}
}

void mtk::curand_fp16::uniform(generator_t &gen, half *const ptr, const std::size_t size) {
	switch (gen.rng_type) {
#define CASE_RNG_TYPE(rng) case rng: generate_kernel<typename mtk::curand_fp16::curand_status_t<rng>::type>\
		<<<gen.num_threads / block_size, block_size, 0, gen.hip_stream>>>\
		(ptr, reinterpret_cast<typename mtk::curand_fp16::curand_status_t<rng>::type*>(gen.status_ptr), size);break
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_MRG32K3A        );
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_XORWOW          );
		CASE_RNG_TYPE(HIPRAND_RNG_PSEUDO_PHILOX4_32_10   );
		default:
			throw std::runtime_error("Unknown pseudo rand algorithm");
#undef CASE_RNG_TYPE
	}
}

void mtk::curand_fp16::destroy(generator_t &gen) {
	hipFree(gen.status_ptr);
}
