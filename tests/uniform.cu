#include "hip/hip_runtime.h"
#include <iostream>
#include <curand_fp16/curand_fp16.hpp>

namespace {
const char* get_curand_rng_name_str(const hiprandRngType_t rng) {
	switch (rng) {
	case HIPRAND_RNG_PSEUDO_XORWOW:
		return "XORWOW";
	case HIPRAND_RNG_PSEUDO_MRG32K3A:
		return "MRG32K3A";
	case HIPRAND_RNG_PSEUDO_PHILOX4_32_10:
		return "PHILOX4_32_10";
	default:
		return "Unknown";
	}
}
} // noname namespace

void test_curand_fp16(
		const std::size_t N,
		hiprandRngType_t rng
		) {
	half* ptr;
	hipMallocManaged(&ptr, sizeof(half) * N);

	mtk::curand_fp16::generator_t generator;
	mtk::curand_fp16::create(generator, rng);
	mtk::curand_fp16::set_seed(generator, 0);

	mtk::curand_fp16::uniform(generator, ptr, N);
	hipDeviceSynchronize();

	double sum = 0;
	for (std::size_t i = 0; i < N; i++) {
		sum += __half2float(ptr[i]);
	}
	const double avg = sum / N;
	double tmp = 0;
	for (std::size_t i = 0; i < N; i++) {
		const auto diff = avg - __half2float(ptr[i]);
		tmp += diff * diff;
	}
	const auto var = tmp / (N - 1);

	std::printf("[%15s] avg = %e [theo = 1/2], var = %e [theo = 1/12]\n", get_curand_rng_name_str(rng), avg, var);

	mtk::curand_fp16::destroy(generator);
	hipFree(ptr);
}

int main() {
	test_curand_fp16(1u << 20, HIPRAND_RNG_PSEUDO_MRG32K3A     );
	test_curand_fp16(1u << 20, HIPRAND_RNG_PSEUDO_XORWOW       );
	test_curand_fp16(1u << 20, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
}
